#include "hip/hip_runtime.h"

#include "src/fastertransformer/kernels/bfloat16_fallback_kenrels.cuh"
#include "src/fastertransformer/kernels/layernorm_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"

namespace fastertransformer {

namespace svpost_relu {

#define sv_eps (1e-5)

// just make output with relu judge

// * Note that typename T is half2 or bfloat2 type
template<typename T, bool IS_OUTPUT, bool IS_BIAS, bool IS_RESIDUAL, bool IS_BETA, int UNROLL_FACTOR>
__global__ void generalAddBiasResidualLayerNormOpt(T* normed_output,
                                                   T* output,
                                                   const T* __restrict bias,
                                                   const T* __restrict residual,
                                                   const T* __restrict gamma,
                                                   const T* __restrict beta,
                                                   int m,
                                                   int n)
{
    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;
    using T1 = typename TypeConverter<T>::Type;  /// half2 --> half

    T type0P = float2type2<T>(.0f);

    T local_sum = float2type2<T>(0.0f);
#pragma unroll
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = blockIdx.x * n + i;
        T val = type0P;

        if (IS_BIAS) {
            val = hadd2(val, ldg(&bias[i]));
        }
        if (IS_RESIDUAL) {
            val = hadd2(val, ldg(&residual[index]));
        }

        if (IS_OUTPUT) {
            val = hadd2(val, output[index]);
        }
        output[index] = val;
        local_sum = hadd2(local_sum, val);
    }

    mean = blockReduceSum((float)(local_sum.x + local_sum.y));

    if (threadIdx.x == 0) {
        s_mean = mean / (n << 1);
    }
    __syncthreads();

    float local_var_sum = 0.0f;
#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        T val = output[blockIdx.x * n + i];
        float diff_1 = (float)(val.x) - s_mean;
        float diff_2 = (float)(val.y) - s_mean;
        local_var_sum += (diff_1 * diff_1 + diff_2 * diff_2);
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / (n << 1) + 1e-5f);
    }
    __syncthreads();

    T mean_2 = float2type2<T>(s_mean);
    T var_2 = float2type2<T>(s_variance);
    T1 t1zero = (T1)0.0f;
#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = blockIdx.x * n + i;
        T val = hmul2(hmul2(hsub2(output[index], mean_2), var_2), ldg(&gamma[i]));
        if (IS_BETA) {
            val = hadd2(val, ldg(&beta[i]));
        }

        if (val.x < t1zero)
            val.x = t1zero;
        if (val.y < t1zero)
            val.y = t1zero;
        normed_output[index] = val;
    }
}

// * Note that typename T is half2 or bfloat2 type
template<typename T, bool IS_OUTPUT, bool IS_BIAS, bool IS_RESIDUAL, bool IS_BETA, int UNROLL_FACTOR>
__global__ void generalAddBiasResidualLayerNormOpt2(T* normed_output,
                                                    T* output,
                                                    const T* __restrict bias,
                                                    const T* __restrict residual,
                                                    const T* __restrict gamma,
                                                    const T* __restrict beta,
                                                    int m,
                                                    int n)
{
    __shared__ float s_mean;
    __shared__ float s_variance;
    float x_sum = 0.0f;
    float x2_sum = 0.0f;
    const int b_offset = blockIdx.x * n;
    using T1 = typename TypeConverter<T>::Type;

#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = b_offset + i;
        float val_1 = 0.0f;
        float val_2 = 0.0f;
        T tmp;

        if (IS_BIAS) {
            tmp = ldg(&bias[i]);
            val_1 += static_cast<float>(tmp.x);
            val_2 += static_cast<float>(tmp.y);
        }
        if (IS_RESIDUAL) {
            tmp = ldg(&residual[index]);
            val_1 += static_cast<float>(tmp.x);
            val_2 += static_cast<float>(tmp.y);
        }

        if (IS_OUTPUT) {
            tmp = output[index];

            val_1 += static_cast<float>(tmp.x);
            val_2 += static_cast<float>(tmp.y);
        }
        tmp.x = float2type<T1>(val_1);
        tmp.y = float2type<T1>(val_2);
        output[index] = tmp;
        x_sum += val_1 + val_2;
        x2_sum += val_1 * val_1 + val_2 * val_2;
    }
    float sums[2];
    sums[0] = x_sum;
    sums[1] = x2_sum;
    blockReduceSumV2<float, 2>(sums);

    if (threadIdx.x == 0) {
        s_mean = sums[0] / (n << 1);
        s_variance = rsqrtf(sums[1] / (n << 1) - s_mean * s_mean + 1e-5f);
    }
    __syncthreads();

    T mean_2 = float2type2<T>(s_mean);
    T var_2 = float2type2<T>(s_variance);
    T1 t1zero = T1(0.0f);
#pragma unroll UNROLL_FACTOR
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        const int index = b_offset + i;
        T val = hmul2(hmul2(hsub2(output[index], mean_2), var_2), ldg(&gamma[i]));
        if (IS_BETA) {
            val = hadd2(val, ldg(&beta[i]));
        }

        if (val.x < t1zero)
            val.x = t1zero;
        if (val.y < t1zero)
            val.y = t1zero;
        normed_output[index] = val;
    }
}

/*******************  invokeAddBiasLayernorm  ***********************/

template<typename T>
__global__ void add_bias_layernorm(T* out, const T* bias, const T* gamma, const T* beta, int n)
{
    int tid = threadIdx.x;
    const int bid = blockIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;

    int idx = bid * n + tid;

    float local_out = (tid < n) ? (float)(out[idx] + ldg(&bias[tid])) : 0.0f;
    mean = blockReduceSum<float>(local_out);
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float diff = (tid < n) ? (local_out - s_mean) : 0.0f;
    variance = blockReduceSum<float>(diff * diff);
    if (threadIdx.x == 0) {
        s_variance = variance / n + 1e-5f;
    }
    __syncthreads();

    if (tid < n) {
        T val = (T)(((local_out - s_mean) * rsqrtf(s_variance)) * (float)(ldg(&gamma[tid])) + (float)(ldg(&beta[tid])));
        out[idx] = val > (T)0.0f ? val : (T)0.0f;
    }
}

template<typename T>
__global__ void
add_bias_layernorm_v2(T* out, const T* __restrict bias, const T* __restrict gamma, const T* __restrict beta, int n)
{
    const int ite = 4;
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int offset = bid * n;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float mean = 0.0f;
    float variance = 0.0f;
    float local_out[ite];

    float sum = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; ++i) {
        int col_id = i * blockDim.x + tid;
        local_out[i] = (col_id < n) ? (float)(out[offset + col_id] + ldg(&bias[col_id])) : 0.0f;
        sum += local_out[i];
    }

    mean = blockReduceSum<float>(sum);
    if (tid == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float var = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id = i * blockDim.x + tid;
        float diff = (col_id < n) ? (local_out[i] - s_mean) : 0.0f;
        var += diff * diff;
    }

    variance = blockReduceSum<float>(var);
    if (tid == 0) {
        s_variance = rsqrtf(variance / n + 1e-5f);
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < ite; ++i) {
        int col_id = i * blockDim.x + tid;
        if (col_id < n) {
            auto val =
                (T)((local_out[i] - s_mean) * s_variance * (float)ldg(&gamma[col_id]) + (float)ldg(&beta[col_id]));
            out[offset + col_id] = val > (T)0.0f ? val : (T)0.0f;
        }
    }
}

#define HALF_LAYERNORM_OPT(UNROLL_FACTOR)                                                                              \
    generalAddBiasResidualLayerNormOpt<T2, false, true, true, true, UNROLL_FACTOR><<<grid, block, 0, stream>>>(        \
        (T2*)out, (T2*)out, (const T2*)bias, (const T2*)out, (const T2*)gamma, (const T2*)beta, m, half_n);

#define HALF_LAYERNORM_OPT2(UNROLL_FACTOR)                                                                             \
    generalAddBiasResidualLayerNormOpt2<T2, false, true, true, true, UNROLL_FACTOR><<<grid, block, 0, stream>>>(       \
        (T2*)out, (T2*)out, (const T2*)bias, (const T2*)out, (const T2*)gamma, (const T2*)beta, m, half_n);

template<typename T>
void invokeAddBiasLayernorm(
    T* out, const T* bias, const T* gamma, const T* beta, int m, int n, hipStream_t stream, int opt_version)
{
    dim3 grid(m);
    if (n % 2 == 0 && std::is_same<T, half>::value && opt_version > 0) {
        int half_n = n / 2;
        int half_n_32 = (half_n + 31) / 32 * 32;
        dim3 block(min(half_n_32, 512));
        int rolls_per_thread = half_n / block.x;
        int unroll_factor = 8;
        while (unroll_factor > rolls_per_thread && unroll_factor > 1) {
            unroll_factor /= 2;
        }
        using T2 = typename TypeConverter<T>::Type;
        if (opt_version == 1) {
            if (unroll_factor == 1) {
                HALF_LAYERNORM_OPT(1);
            }
            else if (unroll_factor == 2) {
                HALF_LAYERNORM_OPT(2);
            }
            else if (unroll_factor == 3) {
                HALF_LAYERNORM_OPT(3);
            }
            else if (unroll_factor == 4) {
                HALF_LAYERNORM_OPT(4);
            }
            else if (unroll_factor == 8) {
                HALF_LAYERNORM_OPT(8);
            }
        }
        else {
            if (unroll_factor == 1) {
                HALF_LAYERNORM_OPT2(1);
            }
            else if (unroll_factor == 2) {
                HALF_LAYERNORM_OPT2(2);
            }
            else if (unroll_factor == 3) {
                HALF_LAYERNORM_OPT2(3);
            }
            else if (unroll_factor == 4) {
                HALF_LAYERNORM_OPT2(4);
            }
            else if (unroll_factor == 8) {
                HALF_LAYERNORM_OPT2(8);
            }
        }
    }
    else {
        /// fp32 oth
        int blockSize = (n + 31) / 32 * 32;
        if (blockSize >= 768) {
            blockSize = ((blockSize / 4) + 31) / 32 * 32;
            add_bias_layernorm_v2<T><<<grid, blockSize, 0, stream>>>(out, bias, gamma, beta, n);
        }
        else {
            add_bias_layernorm<T><<<grid, blockSize, 0, stream>>>(out, bias, gamma, beta, n);
        }
    }
}

#undef HALF_LAYERNORM_OPT
#undef HALF_LAYERNORM_OPT2

template void invokeAddBiasLayernorm<float>(float* out,
                                            const float* bias,
                                            const float* gamma,
                                            const float* beta,
                                            int m,
                                            int n,
                                            hipStream_t stream,
                                            int opt_version);

template void invokeAddBiasLayernorm<half>(half* out,
                                           const half* bias,
                                           const half* gamma,
                                           const half* beta,
                                           int m,
                                           int n,
                                           hipStream_t stream,
                                           int opt_version);
#ifdef ENABLE_BF16
template void invokeAddBiasLayernorm<__hip_bfloat16>(__hip_bfloat16* out,
                                                    const __hip_bfloat16* bias,
                                                    const __hip_bfloat16* gamma,
                                                    const __hip_bfloat16* beta,
                                                    int m,
                                                    int n,
                                                    hipStream_t stream,
                                                    int opt_version);
#endif
}  // namespace svpost_relu

}  // namespace fastertransformer
