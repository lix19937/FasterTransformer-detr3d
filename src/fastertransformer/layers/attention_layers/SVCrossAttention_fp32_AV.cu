#include "hip/hip_runtime.h"
/**************************************************************
 * @Copyright: 2021-2022 Copyright SAIC
 * @Author: lijinwen
 * @Date: 2022-08-14 18:55:58
 * @Last Modified by: lijinwen
 * @Last Modified time: 2022-08-14 18:56:40
 **************************************************************/

#include "SVCrossAttentionLayer.h"

#include "src/fastertransformer/kernels/layernorm_kernels.h"
#include "src/fastertransformer/models/sv/helper_file.h"

namespace fastertransformer {
namespace avt {
namespace fp32 {

__forceinline__ __device__ bool within_bounds_2d(const int h, const int w, const int H, const int W)
{
    return h >= 0 && w >= 0 && h < H && w < W;
}

__global__ void TransposeAndNormKernel(float* __restrict__ out,
                                       const float* __restrict__ in,
                                       const float* __restrict__ range,
                                       const int height,
                                       const int width)
{
    const int BLOCK_DIM = 32;
    __shared__ float block[BLOCK_DIM][BLOCK_DIM + 1];

    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
    if (xIndex < width && yIndex < height) {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = __ldg(&in[index_in]);
    }

    __syncthreads();

    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if (xIndex < height && yIndex < width) {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = block[threadIdx.x][threadIdx.y] * __ldg(&range[yIndex + 3]) + __ldg(&range[yIndex]);
    }
    else if (xIndex < height && yIndex == width) {
        unsigned int index_out = yIndex * height + xIndex;
        out[index_out] = 1;
    }
}

__global__ void ReferencePointsCamAndMaskPermuteKernel(uint8_t* __restrict__ mask,
                                                       float* __restrict__ rpc,
                                                       const float* __restrict__ rpc_matmuled,
                                                       const float* __restrict__ pol_datas,
                                                       const float* __restrict__ cxy_cropxseyse_oxy,
                                                       const int* __restrict__ img_shape,
                                                       const unsigned int NC,
                                                       const unsigned int w,
                                                       const unsigned int L)
{
    int l_idx = threadIdx.x;  // map to L, col direct
    int nc_idx = blockIdx.x;  // map to NC, row direct

    /// 4 is fixed, because width of reference_points is 3, and exp 1 dim
    /// for [1, NC, 4, L] of rpc_matmuled   [:,:,2:3,:]
    unsigned int dim_acc[]{0, 4 * L, L, 1};
    auto idx_23 = nc_idx * dim_acc[1] + 2 * dim_acc[2] + l_idx;
    auto idx_01 = nc_idx * dim_acc[1] + 0 * dim_acc[2] + l_idx;
    auto idx_12 = nc_idx * dim_acc[1] + 1 * dim_acc[2] + l_idx;

    auto reference_points_cam_23 = __ldg(&rpc_matmuled[idx_23]);
    auto reference_points_cam_01 = __ldg(&rpc_matmuled[idx_01]);
    auto reference_points_cam_12 = __ldg(&rpc_matmuled[idx_12]);

    float eps = 1e-5f;
    bool corners_flag = reference_points_cam_23 > 0;
    bool _mask = reference_points_cam_23 > eps;

    float div = max(reference_points_cam_23, 0.01);
    float m = reference_points_cam_01 / div;
    float n = reference_points_cam_12 / div;
    float r = sqrtf(m * m + n * n);
    float theta = corners_flag ? atanf(r) : 3.14 - atanf(r);

    // [1, NC, 1, L] * [1, 4, 1, 1] (will bc)
    float theta_d = /**/ theta * pol_datas[nc_idx * 5]          /* first line */
                    + theta * theta * pol_datas[nc_idx * 5 + 1] /* second line */
                    + theta * theta * theta * pol_datas[nc_idx * 5 + 2]
                    + theta * theta * theta * theta * pol_datas[nc_idx * 5 + 3]
                    + theta * theta * theta * theta * theta * pol_datas[nc_idx * 5 + 4];

    float cdist = r > eps ? theta_d / r : 1;
    // [1, NC, 2, L] *  [1, NC, 1, L]
    m = m * cdist;
    n = n * cdist;

    // center_xy = cxy_cropxseyse_oxy[...,:2]
    // crop_cfg_inputs = cxy_cropxseyse_oxy[...,2:6]  #crop_x_start crop_x_end crop_y_start crop_y_end
    // ori_inputs = cxy_cropxseyse_oxy[...,6:] # [1, 4, 2]

    // #  [1, 4, 1]   -    [1, 4, 512] + [1, 4, 1]
    // reference_points_cam[..., 0,:] = ori_inputs[...,0:1] - (reference_points_cam[..., 0,:] + center_xy[..., 0:1])
    // reference_points_cam[..., 1,:] = ori_inputs[...,1:2] - (reference_points_cam[..., 1,:] + center_xy[..., 1:2])
    m = cxy_cropxseyse_oxy[nc_idx * 8 + 6] - (m + cxy_cropxseyse_oxy[nc_idx * 8]);
    n = cxy_cropxseyse_oxy[nc_idx * 8 + 7] - (n + cxy_cropxseyse_oxy[nc_idx * 8 + 1]);

    //                    # [1,4] * [1,4]
    // crop_x_start = crop_cfg_inputs[:, :, 0] * ori_inputs[:, :, 0]
    // crop_x_end   = crop_cfg_inputs[:, :, 1] * ori_inputs[:, :, 0]
    // crop_y_start = crop_cfg_inputs[:, :, 2] * ori_inputs[:, :, 1]
    // crop_y_end   = crop_cfg_inputs[:, :, 3] * ori_inputs[:, :, 1]
    auto crop_x_start = cxy_cropxseyse_oxy[nc_idx * 8 + 2] * cxy_cropxseyse_oxy[nc_idx * 8 + 6];
    auto crop_x_end = cxy_cropxseyse_oxy[nc_idx * 8 + 3] * cxy_cropxseyse_oxy[nc_idx * 8 + 6];
    auto crop_y_start = cxy_cropxseyse_oxy[nc_idx * 8 + 4] * cxy_cropxseyse_oxy[nc_idx * 8 + 7];
    auto crop_y_end = cxy_cropxseyse_oxy[nc_idx * 8 + 5] * cxy_cropxseyse_oxy[nc_idx * 8 + 7];

    // auto scale_x = img_shape[1] / (crop_x_end - crop_x_start);
    // auto scale_y = img_shape[0] / (crop_y_end - crop_y_start);

    //  // [1, 4, 512] -  [1, 4, 1]
    // m = (m - crop_x_start) * scale_x / img_shape[1];
    // n = (n - crop_y_start) * scale_y / img_shape[0];
    //////////////////////////////////////////////////////////////////////

    // [1, 4, 512] -  [1, 4, 1]
    m = (m - crop_x_start) / (crop_x_end - crop_x_start);
    n = (n - crop_y_start) / (crop_y_end - crop_y_start);

    /// from  [1, NC, 2, L] permute to [1, NC, L, 2], rpc restore layout
    unsigned int dim_acc_d[]{0, L * 2, 2, 1};
    auto idx_01_d = nc_idx * dim_acc_d[1] + l_idx * dim_acc_d[2];
    auto idx_12_d = nc_idx * dim_acc_d[1] + l_idx * dim_acc_d[2] + 1;

    /// from [1, NC, L, 1] permute to [1, L, NC, 1], mask restore layout
    unsigned int dim_acc_e[]{0, NC, 1, 1};
    auto idx_01_e = l_idx * dim_acc_e[1] + nc_idx * dim_acc_e[2];

    m = m + m - 1.f;  //(m - 0.5) * 2;
    n = n + n - 1.f;  //(n - 0.5) * 2;
    rpc[idx_01_d] = m;
    rpc[idx_12_d] = n;

    if (_mask) {
        if (m < 1. && m > -1. && n < 1. && n > -1.) {
            mask[idx_01_e] = 1;  // true
        }
        else {
            mask[idx_01_e] = 0;  // false
        }
    }
    else {
        mask[idx_01_e] = 0;
    }
}

__forceinline__ __device__ float grid_sampler_compute_source_index(const float coord, const int size)
{
    return ((coord + 1.f) * size - 1) * 0.5f;
}

__global__ void
grid_sampler_2d_forward_kernel(const int nthreads,
                               const float* __restrict__ input,
                               const int inp_N,
                               const int inp_C,
                               const int inp_H,
                               const int inp_W,
                               const float* __restrict__ grid,
                               const int grid_H,
                               const int grid_W,
                               float* __restrict__ output /*N = inp_N, C = inp_C, H = grid_H, W = grid_W*/,
                               const int stack_sz,
                               const int stack_idx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nthreads)
        return;

    int C = inp_C;
    int out_H = grid_H;
    int out_W = grid_W;

    int inp_sW = 1;
    int inp_sH = inp_sW * inp_W;
    int inp_sC = inp_sH * inp_H;
    int inp_sN = inp_sC * inp_C;

    int grid_sCoor = 1;
    int grid_sW = grid_sCoor * 2;
    int grid_sH = grid_sW * grid_W;
    int grid_sN = grid_sH * grid_H;  /// grid size : N C H 2

    int out_sW = 1;
    int out_sH = grid_W;  ///  eq: out_sW * grid_W
    int out_sC = out_sH * grid_H;
    int out_sN = out_sC * inp_C;

    int CH = C * out_H;

    //#pragma unroll
    // for (; i < nthreads; i += blockDim.x * gridDim.x)
    {
        const int w = i % out_W;
        const int h = (i / out_W) % out_H;
        const int n = i / (out_H * out_W);
        const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

        // get the corresponding input x, y co-ordinates from grid
        float ix = __ldg(&grid[grid_offset]);
        float iy = __ldg(&grid[grid_offset + grid_sCoor]);
        ix = grid_sampler_compute_source_index(ix, inp_W);
        iy = grid_sampler_compute_source_index(iy, inp_H);

        // get NE, NW, SE, SW pixel values from (x, y)
        int ix_nw = static_cast<int>(::floor(ix));
        int iy_nw = static_cast<int>(::floor(iy));

        int ix_ne = ix_nw + 1;
        int iy_ne = iy_nw;
        int ix_sw = ix_nw;
        int iy_sw = iy_nw + 1;
        int ix_se = ix_nw + 1;
        int iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        float nw = (ix_se - ix) * (iy_se - iy);
        float ne = (ix - ix_sw) * (iy_sw - iy);
        float sw = (ix_ne - ix) * (iy - iy_ne);
        float se = (ix - ix_nw) * (iy - iy_nw);

        /// calculate bilinear weighted pixel value and set output pixel
        auto inp_ptr_NC = input + n * inp_sN;
        /// auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
        auto in_offset = n * out_sN + h * out_sH + w * out_sW;

        float sum;

        // int offset1 = iy_nw * inp_sH + ix_nw * inp_sW;
        // int offset2 = iy_ne * inp_sH + ix_ne * inp_sW;
        // int offset3 = iy_sw * inp_sH + ix_sw * inp_sW;
        // int offset4 = iy_se * inp_sH + ix_se * inp_sW;

        int mul1 = iy_nw * inp_sH;  // iy_nw == iy_ne
        int mul2 = ix_nw * inp_sW;  // ix_nw == ix_sw
        int mul3 = iy_sw * inp_sH;  // iy_sw == iy_se
        int mul4 = ix_ne * inp_sW;  // ix_ne == ix_se

        int offset1 = mul1 + mul2;
        int offset2 = mul1 + mul4;
        int offset3 = mul3 + mul2;
        int offset4 = mul3 + mul4;

#pragma unroll
        for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, in_offset += out_sC) {
            sum = 0.f;
            if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
                sum += inp_ptr_NC[offset1] * nw;
            }
            if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
                sum += inp_ptr_NC[offset2] * ne;
            }
            if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
                sum += inp_ptr_NC[offset3] * sw;
            }
            if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
                sum += inp_ptr_NC[offset4] * se;
            }

            ///             inp_N, inp_C, grid_H, grid_W  [6, 256, L, 1],
            /// so simplify to [nc, 256, L] --> [nc, 256x L]
            {
                int c_idx = in_offset / CH;
                int hw_idx = in_offset % CH;
                int out_offset = (hw_idx * inp_N + c_idx) * stack_sz + stack_idx;
                output[out_offset] = sum;
            }
        }
    }
}

__forceinline__ __device__ void convertchw_maxpool2(
    const size_t idx, const size_t area, const size_t w, const float* __restrict__ input, float* __restrict__ value)
{
    auto AREA = area << 2;  /// 4 * area, W = 2 * w, before pool
    auto raw_idx = (idx / area * AREA) + ((idx % area / w) << 1) * (w << 1) + (idx % area % w << 1);

    *value = __ldg(&input[raw_idx]);
}

__global__ void grid_sampler_2d_forward_kernel_n6c256gh512gw1_maxpool(
    const int nthreads,
    const float* __restrict__ input,
    const int inp_N,
    const int inp_C,  // 256
    const int inp_H,
    const int inp_W,
    const float* __restrict__ grid /* __restrict__ reference_points_cam after norm */,
    const int grid_H,  // 512
    const int grid_W,  // 1
    float* __restrict__ output /* N = inp_N, C = inp_C, H = grid_H, W = grid_W*/,
    const int stack_sz,
    const int stack_idx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nthreads)
        return;

    int C = inp_C;
    int out_H = grid_H;
    int out_W = grid_W;

    int inp_sW = 1;
    int inp_sH = inp_sW * inp_W;
    int inp_sC = inp_sH * inp_H;
    int inp_sN = inp_sC * inp_C;

    int grid_sCoor = 1;
    int grid_sW = grid_sCoor * 2;
    int grid_sH = grid_sW * grid_W;
    int grid_sN = grid_sH * grid_H; /* grid size : N C H 2 */

    int out_sW = 1;
    int out_sH = grid_W;  ///  eq: out_sW * grid_W
    int out_sC = out_sH * grid_H;
    int out_sN = out_sC * inp_C;

    int CH = C * out_H;

    //#pragma unroll
    // for (; i < nthreads; i += blockDim.x * gridDim.x)
    {
        const int w = i % out_W;
        const int h = (i / out_W) % out_H;
        const int n = i / (out_H * out_W);
        const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

        // get the corresponding input x, y co-ordinates from grid
        float ix = __ldg(&grid[grid_offset]);
        float iy = __ldg(&grid[grid_offset + grid_sCoor]);
        ix = grid_sampler_compute_source_index(ix, inp_W);
        iy = grid_sampler_compute_source_index(iy, inp_H);

        // get NE, NW, SE, SW pixel values from (x, y)
        int ix_nw = static_cast<int>(::floor(ix));
        int iy_nw = static_cast<int>(::floor(iy));

        int ix_ne = ix_nw + 1;
        int iy_ne = iy_nw;
        int ix_sw = ix_nw;
        int iy_sw = iy_nw + 1;
        int ix_se = ix_nw + 1;
        int iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        float nw = (ix_se - ix) * (iy_se - iy);
        float ne = (ix - ix_sw) * (iy_sw - iy);
        float sw = (ix_ne - ix) * (iy - iy_ne);
        float se = (ix - ix_nw) * (iy - iy_nw);

        /// calculate bilinear weighted pixel value and set output pixel
        auto inp_ptr_NC = n * inp_sN;
        /// auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
        auto in_offset = n * out_sN + h * out_sH + w * out_sW;

        // int offset1 = iy_nw * inp_sH + ix_nw * inp_sW;
        // int offset2 = iy_ne * inp_sH + ix_ne * inp_sW;
        // int offset3 = iy_sw * inp_sH + ix_sw * inp_sW;
        // int offset4 = iy_se * inp_sH + ix_se * inp_sW;

        int mul1 = iy_nw * inp_sH;  // iy_nw == iy_ne
        int mul2 = ix_nw * inp_sW;  // ix_nw == ix_sw
        int mul3 = iy_sw * inp_sH;  // iy_sw == iy_se
        int mul4 = ix_ne * inp_sW;  // ix_ne == ix_se

        int offset1 = mul1 + mul2;
        int offset2 = mul1 + mul4;
        int offset3 = mul3 + mul2;
        int offset4 = mul3 + mul4;
        const size_t area = inp_H * inp_W;
        float sum, value;

#pragma unroll
        for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, in_offset += out_sC) {
            sum = 0.f;
            if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
                convertchw_maxpool2(inp_ptr_NC + offset1, area, inp_W, input, &value);
                sum += value * nw;
            }
            if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
                convertchw_maxpool2(inp_ptr_NC + offset2, area, inp_W, input, &value);
                sum += value * ne;
            }
            if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
                convertchw_maxpool2(inp_ptr_NC + offset3, area, inp_W, input, &value);
                sum += value * sw;
            }
            if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
                convertchw_maxpool2(inp_ptr_NC + offset4, area, inp_W, input, &value);
                sum += value * se;
            }

            ///             inp_N, inp_C, grid_H, grid_W  [6, 256, L, 1],
            /// so simplify to [nc, 256, L] --> [nc, 256x L]
            {
                int c_idx = in_offset / CH;
                int hw_idx = in_offset % CH;
                int out_offset = (hw_idx * inp_N + c_idx) * stack_sz + stack_idx;
                output[out_offset] = sum;
            }
        }
    }
}

///         attention_weights = attention_weights + attention_weights_bias
///         attention_weights = attention_weights.sigmoid() * mask
__global__ void AddBiasAttentionWeightsSigmoidMaskKernel(float* __restrict__ out,
                                                         const float* __restrict__ fc_attention_out,
                                                         const float* __restrict__ fc_attention_bias,
                                                         const uint8_t* __restrict__ fs_mask,
                                                         const int bias_len)
{
    /// blockDim.x is l
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;  // nc *1*4, l
    auto fs_mask_idx = idx / 4;                        // idx/4  !!!    [l, nc]
    auto bias_idx = idx % bias_len;                    // bias_len [1, nc*1*4]

    if (__ldg(&fs_mask[fs_mask_idx]) == 0) {
        out[idx] = 0.f;
    }
    else {
        float x = __ldg(&fc_attention_out[idx]) + __ldg(&fc_attention_bias[bias_idx]);
        out[idx] = 1.f / (1.f + exp(-x));
    }
}

__global__ void MulAndReducesumKernel(float* __restrict__ reduce_output,
                                      const float* __restrict__ fs_output,
                                      const float* __restrict__ fc_attention_s_m_out,
                                      const int fc_attention_s_m_out_len,
                                      const int scale,
                                      const int reduce_num)
{
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;
    auto m_idx = idx % fc_attention_s_m_out_len;
    auto tid = threadIdx.x;
    extern __shared__ float shm[];

    /// reduce_num = nc*m*k, if scale =1, blockDim.x = reduce_num
    shm[tid] = __ldg(&fs_output[idx]) * __ldg(&fc_attention_s_m_out[m_idx]);
    __syncthreads();

    // #pragma unroll
    //     for (unsigned int s = reduce_num / 2; s > 0; s /= 2) {
    //         if (tid < s) {
    // #pragma unroll
    //             for (int offset = 0; offset < blockDim.x; offset += reduce_num) {
    //                 shm[tid + offset] += shm[tid + s + offset];
    //             }
    //         }
    //         __syncthreads();
    //     }

    //     auto bid = blockIdx.x * scale;
    // #pragma unroll
    //     for (unsigned int i = 0; i < blockDim.x; i += reduce_num) {
    //         if (tid == i) {
    //             // transpose, [ch,  L] --> [L, ch]
    //             unsigned int out_idx = bid + i / reduce_num;
    //             unsigned int r = out_idx / 512;                          ///  out_idx / L
    //             unsigned int c = out_idx & 511;                          ///  out_idx % L
    //             reduce_output[(c * 256) + r] = shm[tid] + shm[tid + 2];  ///  c*ch + r
    //         }
    //     }

    if (tid == 0) {
        for (int i = 1; i < 16; ++i) {
            shm[0] += shm[i];
        }
        unsigned int r = blockIdx.x / 512;      ///  out_idx / L
        unsigned int c = blockIdx.x & 511;      ///  out_idx % L
        reduce_output[(c * 256) + r] = shm[0];  ///  c*ch + r
    }
}

///   output = output * attention_weights
/// torch.Size([1, 256, 512, 4, 1, 4]) torch.Size([1, 1, 512, 4, 1, 4])
/// note: output is fs_output;  attention_weights = attention_weights.sigmoid() * mask
///  fc_attention_s_m_out means out of `attention_weights.sigmoid() * mask`
///  here we can Dimensionality reduction, [CH, L*NC*1*4] x [1, L*NC*1*4]
__global__ void output_x_attention_weights_kernel(float* out,
                                                  const float* fs_output,
                                                  const float* fc_attention_s_m_out,
                                                  const int fc_attention_s_m_out_len)
{
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;
    auto m_idx = idx % fc_attention_s_m_out_len;

    out[idx] = __ldg(&fs_output[idx]) * __ldg(&fc_attention_s_m_out[m_idx]);
}

__global__ void IsigmoidAddBiasKernel(float* out, const float* __restrict__ rp /* reference_points */)
{
    auto idx = threadIdx.x + blockIdx.x * blockDim.x;
    const float eps = 1.e-5;

    float x = __ldg(&rp[idx]);
    x = min(max(x, 0.f), 1.f);
    auto x1 = max(x, eps);
    auto x2 = max(1.f - x, eps);
    out[idx] = log(x1 / x2);
}

void ReferencePointsNorm(float* rp_norm,        /* reference_points after norm [L, 3](memory is L*4)  -->  [4, L] */
                         const float* rp,       /* reference_points`s shape B=1, from IN */
                         const float* pc_range, /* from Attri */
                         const int in_h,        /* L */
                         const int in_w,        /* 3 */
                         hipStream_t stream)
{
    const int block_w = 32, block_h = 32;
    dim3 grid((in_w + block_w - 1) / block_w, (in_h + block_h - 1) / block_h);
    dim3 block(block_w, block_h);
    TransposeAndNormKernel<<<grid, block, 0, stream>>>(rp_norm, rp, pc_range, in_h, in_w);
}

// l2i * rp
// [NC*4, 4] * [4, L] -->  [NC*4, L] or [1, NC, 4, L]  rpc_matmuled = l2i_norm * rp_norm
void L2IxReferencePoints(float* rpc_matmuled,  /* reference_points_cam after rp matmul; [1, NC, 4, L] */
                         const float* l2i,     /* lidar2img [NC*4, 4] */
                         const float* rp_norm, /* rp has norm; [4, L] */
                         int m,                /* nc * 4 */
                         int k,                /* 4 */
                         int n,                /* l */
                         cublasMMWrapper* cublas_wrapper,
                         hipStream_t stream)
{
    cublas_wrapper->Gemm(HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, rp_norm, n, l2i, k, rpc_matmuled, n, 1.f, 0.f);
}

void ReferencePointsCamAndMask(uint8_t* mask,             /* mask, last out; [1, L, NC, 1] */
                               float* rpc_norm,           /* reference_points_cam after norm; [1, NC, L, 2] */
                               const float* rpc_matmuled, /* reference_points_cam after torch.matmul; [1, NC, 4, L]*/
                               const float* pol_datas,
                               const float* cxy_cropxseyse_oxy,
                               const int* img_shape,
                               const unsigned int nc,
                               const unsigned int w, /* 4 */
                               const unsigned int l,
                               hipStream_t stream)
{
    /// here we make sure L < 1024, and w == 4, Here will be improve
    dim3 grid(nc, 1);
    dim3 block(l, 1);
    ReferencePointsCamAndMaskPermuteKernel<<<grid, block, 0, stream>>>(
        mask, rpc_norm, rpc_matmuled, pol_datas, cxy_cropxseyse_oxy, img_shape, nc, w, l);
}

void BatchedBilinearGridSample(float* sampled_feats,  /* sampled_feats, last out; [CH, L, NC, 1, 4] */
                               const float* rpc_norm, /* reference_points_cam after norm; [1, NC, L, 2] */
                               const float* mlvl_feats[],
                               const std::vector<std::vector<size_t>>& mlvl_feats_dims, /* [4]; [ NC, Ch, _, _] */
                               const int seq_len,                                       /* num_cam, num_query */
                               hipStream_t stream)
{
    const int inp_N = mlvl_feats_dims[0][0];
    const int inp_C = mlvl_feats_dims[0][1];

    const int grid_H = seq_len;
    const int grid_W = 1;              /// here we make sure grid_W == 1 !!!
    const int count = inp_N * grid_H;  /// inp_N * grid_H * grid_W
    const int stack_sz = mlvl_feats_dims.size();

    dim3 block(512 / 2);
    dim3 grid((count + block.x - 1) / block.x);

    for (int i = 0; i < stack_sz; ++i) {
        const int inp_H = mlvl_feats_dims[i][2];
        const int inp_W = mlvl_feats_dims[i][3];

        grid_sampler_2d_forward_kernel<<<grid, block, 0, stream>>>(count,
                                                                   mlvl_feats[i],
                                                                   inp_N,
                                                                   inp_C,
                                                                   inp_H,
                                                                   inp_W,
                                                                   rpc_norm,
                                                                   grid_H,
                                                                   grid_W,
                                                                   sampled_feats,
                                                                   4,  // stack_sz,
                                                                   i);
        /// last loop, then add maxpool
        if (i + 1 == stack_sz) {
            grid_sampler_2d_forward_kernel_n6c256gh512gw1_maxpool<<<grid, block, 0, stream>>>(count,
                                                                                              mlvl_feats[i],
                                                                                              inp_N,
                                                                                              inp_C,
                                                                                              inp_H / 2,
                                                                                              inp_W / 2,
                                                                                              rpc_norm,
                                                                                              grid_H,
                                                                                              grid_W,
                                                                                              sampled_feats,
                                                                                              4,  // stack_sz,
                                                                                              i + 1);
        }
    }
}

// [seq_len, embed_dims] * [embed_dims, 24] --> [seq_len, 24] -> [seq_len, nc, 1, 4]
void AttentionWeightsFc(float* attention_weights,
                        const float* query_embbed, /* attention_weights = self.attention_weights(query) */
                        const float* fc_attention_weights,
                        const int m,
                        const int k,
                        const int n,
                        cublasMMWrapper* cublas_wrapper,
                        hipStream_t stream)
{
    cublas_wrapper->Gemm(
        HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, fc_attention_weights, n, query_embbed, k, attention_weights, n, 1.f, 0.f);
}

void TwoMulReduceSum(float* reduce_output,           /* last output */
                     float* fc_attention_s_m_out,    /* attention_weights.sigmoid() * mask */
                     const float* fc_attention_out,  /* from AttentionWeightsFc output */
                     const float* fc_attention_bias, /* AttentionWeightsFc which bias here */
                     const float* fs_output,
                     const uint8_t* fs_mask, /* from feature sampling`s output */
                     const int ch,
                     const int l,
                     const int nc,
                     const int m,
                     const int k,
                     hipStream_t stream)
{
    dim3 block(l);
    dim3 grid(nc * m * k);

    AddBiasAttentionWeightsSigmoidMaskKernel<<<grid, block, 0, stream>>>(
        fc_attention_s_m_out, fc_attention_out, fc_attention_bias, fs_mask, grid.x);

    // output_x_attention_weights_kernel<<<ch * l, nc * m * k, 0, stream>>>(
    //     reduce_output, fs_output, fc_attention_s_m_out, l * nc * m * k);

    {
        const int scale = 1;
        dim3 block(nc * m * k * scale);
        dim3 grid(ch * l / scale);
        MulAndReducesumKernel<<<grid, block, block.x * sizeof(float), stream>>>(
            reduce_output, fs_output, fc_attention_s_m_out, l * nc * m * k, scale, nc * m * k);
    }
}

void OutputProjFc(float* output,
                  const float* reduce_output,
                  const float* output_proj_weight,
                  const int m,
                  const int k,
                  const int n,
                  cublasMMWrapper* cublas_wrapper,
                  hipStream_t stream)
{
    cublas_wrapper->Gemm(
        HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, output_proj_weight, n, reduce_output, k, output, n, 1.f, 0.f);
}

void PositionEncoder(float* pos_feat, /* [seq_len, embed_dims] */
                     float* buf[],
                     const float* rp, /* [seq_len, 3] */
                     const float* weights[],
                     int m, /* seq_len */
                     int k, /* 3 */
                     int n, /* embed_dims */
                     cublasMMWrapper* cublas_wrapper,
                     hipStream_t stream)
{
    IsigmoidAddBiasKernel<<<k, m, 0, stream>>>(buf[0], rp);

    // [L, 3] * [3, E_d] --> [L, E_d]   + [1, E_d] bias
    cublas_wrapper->Gemm(HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, weights[0], n, buf[0], k, buf[1], n, 1.f, 0.f);

    // T* out, const T* bias, const T* gamma, const T* beta, int m, int n, hipStream_t stream
    svpost_relu::invokeAddBiasLayernorm(buf[1], weights[1], weights[2], weights[3], m, n, stream);

    // [L, E_d] * [E_d, E_d] --> [L, E_d]  + [1, E_d] bias
    k = n;
    cublas_wrapper->Gemm(HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, weights[4], n, buf[1], k, pos_feat, n, 1.f, 0.f);

    svpost_relu::invokeAddBiasLayernorm(pos_feat, weights[5], weights[6], weights[7], m, n, stream);
}
}  // namespace fp32
}  // namespace avt

template<>
void SVCrossAttentionLayer<float>::__forward_fp32_AV_branch(std::vector<fastertransformer::Tensor>* output_tensors,
                                                            const std::vector<fastertransformer::Tensor>* input_tensors,
                                                            const CAttentionWeight<float>* weights)
{
    const int nc = num_cam_, seq_len = num_query_, embed_dims = hidden_units_, ch = input_tensors->at(2).shape[1];

    /// OUT
    auto d_output_prj = (float*)output_tensors->at(0).data; /* [L, embed_dims] */
    auto d_pos_feat = (float*)output_tensors->at(1).data;   /* [L, embed_dims] */

    /// IN
    auto in_query_embbed = (const float*)input_tensors->at(0).data; /* [L, embed_dims]  has added with query_pos */
    auto in_rp = (const float*)input_tensors->at(1).data;           /* [1, L, 3] reference_points */
    const float* in_x[]{(const float*)input_tensors->at(2).data,
                        (const float*)input_tensors->at(3).data,
                        (const float*)input_tensors->at(4).data};
    const std::vector<std::vector<size_t>> in_x_shapes{
        input_tensors->at(2).shape, input_tensors->at(3).shape, input_tensors->at(4).shape}; /* [NC, Ch, _, _] */

    auto in_l2i = (const float*)input_tensors->at(5).data;                /* [1, NC, 4, 4] */
    auto in_pol_datas = (const float*)input_tensors->at(6).data;          /* [4, 5] */
    auto in_cxy_cropxseyse_oxy = (const float*)input_tensors->at(7).data; /* [4, 8] */

    /// wieghts
    const auto w_fc_attention_weight = weights->attention_weights.kernel;
    const auto w_fc_attention_bias = weights->attention_weights.bias;
    const auto w_output_proj_weight = weights->output_proj.kernel;
    const float* w_pencoder_weights[]{weights->position_encoder_fc1.kernel,
                                      weights->position_encoder_fc1.bias,
                                      weights->position_encoder_ln1.kernel,
                                      weights->position_encoder_ln1.bias,
                                      weights->position_encoder_fc2.kernel,
                                      weights->position_encoder_fc2.bias,
                                      weights->position_encoder_ln2.kernel,
                                      weights->position_encoder_ln2.bias};

    avt::fp32::ReferencePointsNorm(d_rp_norm_,                /* rp after norm; [1, 4, num_query] */
                                   in_rp,                     /* rp from node in const; [1, num_query, 3] */
                                   (const float*)d_pc_range_, /* from node attribute */
                                   seq_len,
                                   3, /* ReferencePoints width 3  */
                                   stream_);

    avt::fp32::L2IxReferencePoints(d_rp_matmuled_,           /* rp after matmul; [1, NC, 4, L] */
                                   (const float*)in_l2i,     /* lidar2img; [1, 6, 4, 4] */
                                   (const float*)d_rp_norm_, /* rp after norm; [4, L] */
                                   nc * 4,
                                   4,
                                   seq_len,
                                   cublas_wrapper_,
                                   stream_);

    avt::fp32::ReferencePointsCamAndMask(d_fs_mask_,  /* last out; [1, L, NC, 1] */
                                         d_rpc_norm_, /* rpc after norm which as grid sample in;  [1, NC, L, 2] */
                                         (const float*)d_rp_matmuled_, /* [1, NC, 4, L] */
                                         (const float*)in_pol_datas,
                                         (const float*)in_cxy_cropxseyse_oxy,
                                         (const int*)d_img_shape_,
                                         nc,
                                         4u,
                                         seq_len,
                                         stream_);

    avt::fp32::BatchedBilinearGridSample(d_fs_output_, /* last out; [1, Ch, L, NC, 1, 4] */
                                         (const float*)d_rpc_norm_,
                                         in_x,
                                         in_x_shapes,
                                         seq_len,
                                         stream_);

    avt::fp32::AttentionWeightsFc(d_attention_weights_output_, /* output; [L, 24] , no bias*/
                                  in_query_embbed,             /* [L, embed_dims] */
                                  w_fc_attention_weight,       /* [embed_dims, 24] */
                                  seq_len,
                                  embed_dims,
                                  nc * 4,
                                  cublas_wrapper_,
                                  stream_);

    avt::fp32::TwoMulReduceSum(d_reduce_output_,            /* output; [L, ch] */
                               d_pencoder_bufs_1_,          /* buffer space [L, nc*1*4] */
                               d_attention_weights_output_, /* from AttentionWeightsFc output */
                               w_fc_attention_bias,         /* AttentionWeightsFc which bias here [1, nc*1*4] */
                               (const float*)d_fs_output_,  /* [1, Ch, L, NC, 1, 4] */
                               (const uint8_t*)d_fs_mask_,  /* [1, L, NC, 1] */
                               ch,
                               seq_len,
                               nc,
                               1,
                               4,
                               stream_);

    avt::fp32::OutputProjFc(d_output_prj, /* last output;  [L, embed_dims] */
                            (const float*)d_reduce_output_,
                            w_output_proj_weight, /* [embed_dims, embed_dims]  */
                            seq_len,
                            embed_dims,
                            embed_dims,
                            cublas_wrapper_,
                            stream_);

    avt::fp32::PositionEncoder(d_pos_feat,       /* last output;  [L, embed_dims] */
                               d_pencoder_bufs_, /* tmp space */
                               in_rp,            /* const input */
                               w_pencoder_weights,
                               seq_len,
                               3,
                               embed_dims,
                               cublas_wrapper_,
                               stream_);
}

template<>
void SVCrossAttentionLayer<float>::__forward_magic_fp32_AV_branch(
    std::vector<fastertransformer::Tensor>* output_tensors,
    const std::vector<fastertransformer::Tensor>* input_tensors,
    const CAttentionWeight<float>* weights,
    const HelperIRPara<float>* helper_weights,
    hipStream_t stream)
{
    stream_ = stream;
    const int nc = num_cam_, seq_len = num_query_, embed_dims = hidden_units_, ch = input_tensors->at(0).shape[1];

    /// OUT
    auto d_output_prj = (float*)output_tensors->at(0).data; /* [L, embed_dims] */

    /// IN
    const float* in_x[]{(const float*)input_tensors->at(0).data,
                        (const float*)input_tensors->at(1).data,
                        (const float*)input_tensors->at(2).data};
    const std::vector<std::vector<size_t>> in_x_shapes{
        input_tensors->at(0).shape, input_tensors->at(1).shape, input_tensors->at(2).shape}; /* [NC, Ch, _, _] */

    auto in_l2i = (const float*)input_tensors->at(3).data;                /* [NC, 4, 4] */
    auto in_pol_datas = (const float*)input_tensors->at(4).data;          /* [4, 5] */
    auto in_cxy_cropxseyse_oxy = (const float*)input_tensors->at(5).data; /* [4, 8] */

    /// weights
    const auto w_fc_attention_bias = weights->attention_weights.bias;
    const auto w_output_proj_weight = weights->output_proj.kernel;

    avt::fp32::L2IxReferencePoints((float*)d_rp_matmuled_,  /* rp after matmul; [1, NC, 4, L] */
                                   (const float*)in_l2i,    /* lidar2img; [1, NC, 4, 4] */
                                   helper_weights->rp_norm, /* rp after norm; [4, L], const */
                                   nc * 4,                  /* l2i_norm h  */
                                   4,                       /* l2i_norm w  */
                                   seq_len,                 /* rp_norm w */
                                   cublas_wrapper_,
                                   stream_);

    // FT_SAVE<float>("ir.ca.fs.rfpcat.log", {4, seq_len}, (float*)helper_weights->rp_norm);
    // FT_SAVE<float>("ca.fs.rfpcammatmul.log", {1, nc, 4, seq_len}, (float*)d_rp_matmuled_);

    avt::fp32::ReferencePointsCamAndMask(d_fs_mask_,  /* last out; [1, L, NC, 1] */
                                         d_rpc_norm_, /* rpc after norm;  [1, NC, L, 1, 2] */
                                         (const float*)d_rp_matmuled_,
                                         (const float*)in_pol_datas,
                                         (const float*)in_cxy_cropxseyse_oxy,
                                         (const int*)d_img_shape_,
                                         nc,
                                         4u,
                                         seq_len,
                                         stream_);

    // reference_points_3d, output, mask = feature_sampling_onnx(
    //     value, reference_points, self.pc_range, kwargs['img_shape'], kwargs['lidar2img'])
    avt::fp32::BatchedBilinearGridSample(d_fs_output_, /* last out; [1, Ch, L, NC, 1, 4] */
                                         (const float*)d_rpc_norm_,
                                         in_x,
                                         in_x_shapes,
                                         seq_len,
                                         stream_);
    // FT_SAVE<float>("ca.fs.rfpcam_norm.log", {nc, seq_len, 1, 2}, (float*)d_rpc_norm_);       // passed
    // FT_SAVE<float>("ca.fs_output.log", {ch, seq_len, nc, 1, 4}, (float*)d_fs_output_);  //
    // FT_SAVE<uint8_t>("ca.fs_mask.log", {seq_len, nc, 1, 1}, (uint8_t*)d_fs_mask_);      // passed

    ///////// helper_weights->attention_weights_output is follow:
    // attention_weights = self.attention_weights(query).view(
    //     bs, 1, num_query, self.num_cams, self.num_points, self.num_levels)

    ///////// d_reduce_output_ is output of last
    /// attention_weights = attention_weights.sigmoid() * mask
    /// output = output * attention_weights
    /// output = output.sum(-1).sum(-1).sum(-1)
    /// output = output.permute(2, 0, 1)
    avt::fp32::TwoMulReduceSum(
        d_reduce_output_,                         /* output; [L, ch] */
        d_pencoder_bufs_1_,                       /* buff space [L, nc*1*4] */
        helper_weights->attention_weights_output, /* AttentionWeightsFc`s output without bias, [L, nc*1*4]  */
        w_fc_attention_bias,                      /* AttentionWeightsFc which bias here [1, nc*1*4] */
        (const float*)d_fs_output_,
        (const uint8_t*)d_fs_mask_,
        ch,
        seq_len,
        nc,
        1,
        4,
        stream_);
    // FT_SAVE<float>("ir.ca.attention_weights.w_fc_attention_bias.log", {1, nc * 1 * 4}, (float*)w_fc_attention_bias);
    // FT_SAVE<float>("ir.ca.attention_weights.out_before_reducesum.log", {seq_len, nc, 1, 4},
    // (float*)d_pencoder_bufs_1_); FT_SAVE<float>("ca.out_after_reducesum.log", {seq_len, ch},
    // (float*)d_reduce_output_);

    /// output = self.output_proj(output)
    /// d_output_prj without `w_output_proj_bias`, but not CA`s out
    /// CA`S out = output_proj`s out + inp_residual + position_encoder`s out
    avt::fp32::OutputProjFc(d_output_prj, /* last output;  [L, embed_dims] */
                            (const float*)d_reduce_output_,
                            w_output_proj_weight, /* [embed_dims, embed_dims]  */
                            seq_len,
                            embed_dims,
                            embed_dims,
                            cublas_wrapper_,
                            stream_);
    // FT_SAVE<float>("ca.out.subbias.log", {seq_len, embed_dims}, (float*)d_output_prj);
}

}  // namespace fastertransformer
